#include "hip/hip_runtime.h"
#include <stdlib.h>  // Standard input/output library
#include <stdio.h>  // Standard library
#include "read_csr.h" // For import matrix into CSR format
#include "read_ellpack.h"  // For import matrix into ELLPACK format store in 1D array.
#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers
#include <math.h> // For abs and max
#include <string.h>

char* default_filename = "result_gpu.csv";
const int ntimes = 5;

void MatrixVectorCSR(int M, int N, const int* IRP, const int* JA,
 const double* AZ, const double* x, double* y);
void MatrixVectorELLPACK(int M, int N, int NNZ, int MAXNZ, const int* JA,
 const double* AZ, const double* x, double* y);
void check_result(int M, double* y_s_c, double* y, double* max_abs_diff, double* max_rel_diff);
void save_result_cuda(char *program_name,      char* matrix_file,          int M, int N,                     int NNZ, int MAZNZ,
                      int cudaXBD,             int cudaYBD,                int cudaXGD,                      int cudaYGD,
                      double time_csr_serial,  double mflops_csr_serial,   double max_abs_diff_csr_serial,   double max_rel_diff_csr_serial,
                      double time_ell_serial,  double mflops_ell_serial,   double max_abs_diff_ell_serial,   double max_rel_diff_ell_serial,
                      double time_csr_gpu,     double mflops_csr_gpu,      double max_abs_diff_csr_gpu,      double max_rel_diff_csr_gpu,
                      double time_ell_1d_gpu,  double mflops_ell_1d_gpu,   double max_abs_diff_ell_1d_gpu,   double max_rel_diff_ell_1d_gpu,
                      double time_ell_2d_gpu,  double mflops_ell_2d_gpu,   double max_abs_diff_ell_2d_gpu,   double max_rel_diff_ell_2d_gpu, 
                      double time_ell_2dt_gpu, double mflops_ell_2dt_gpu,  double max_abs_diff_ell_2dt_gpu,  double max_rel_diff_ell_2dt_gpu);

__global__ void gpuMatrixVectorCSR(const int XBD, const int YBD, int M, int N, const int* IRP,
 const int* JA, const double* AZ, const double* x, double* y);
__global__ void gpuMatrixVectorELL(const int XBD, const int YBD, int M, int N, int NNZ, int MAXNZ,
 const int* JA, const double* AZ, const double* x, double* y);
__global__ void gpuMatrixVectorELL_2d(const int XBD, const int YBD, int M, int N, int NNZ, int MAXNZ,
 const int* JA, const double* AZ, const double* x, double* y, size_t pitch_JA, size_t pitch_AZ);
__global__ void gpuMatrixVectorELL_2dt(const int XBD, const int YBD, int M, int N, int NNZ, int MAXNZ,
 const int* JAt, const double* AZt, const double* x, double* y, size_t pitch_JA, size_t pitch_AZ);

int main(int argc, char** argv) 
{
  char *program_name = argv[0];
  //printf("Run from file %s\n", program_name);
  char* matrix_file;
  int XBD=128;  // 2d block dimension
  int YBD=8;  // 2d block dimension 
  if (argc == 2) {
    matrix_file = argv[1];
  } else if(argc == 4){
    matrix_file = argv[1];
    XBD = atoi(argv[2]);
    YBD = atoi(argv[3]);
  } else if (argc == 5) {
    matrix_file = argv[1];
    XBD = atoi(argv[2]);
    YBD = atoi(argv[3]);
    default_filename = argv[4];
  } else {
    printf(" Usage: %s matrixFile.mtx [XBD] [YBD] \n", argv[0]);
    return -1;
  }
  printf("---------------------------------------------------------------------\n");
  printf("Run from file: %s, reading matrix: %s, XBD: %d, YBD: %d\n", program_name, matrix_file, XBD, YBD);
  
  // Create the CUDA SDK timer.
  StopWatchInterface* timer = 0;
  sdkCreateTimer(&timer);

  // ======================= Import Matrix Data ======================= //

  // Save matrix file into memory in CSR format.
  struct csr_matrix matrix_csr;
  int ret_code;
  ret_code = read_csr_matrix(matrix_file, &matrix_csr);
  if (ret_code != 0) {
    printf(" Failed to read matrix file\n");
    return ret_code;
  }
  printf("finish loading matrix into CSR format\n");

  // Save matrix file into memory in ELLPACK format.
  struct ellpack_matrix matrix_ellpack;
  ret_code = read_ellpack_matrix(matrix_file, &matrix_ellpack);
  if (ret_code != 0) {
    printf(" Failed to read matrix file\n");
    return ret_code;
  }
  printf("finish loading matrix into 1D ELLPACK format\n");

  //transpose matrix JA and AZ from 1D ELLPACK format >. to achieve row-wise  
  int* JAt = (int*) malloc(matrix_ellpack.M * matrix_ellpack.MAXNZ * sizeof(int));
  double* AZt = (double*) malloc(matrix_ellpack.M * matrix_ellpack.MAXNZ * sizeof(double));
  
  for (int i = 0; i < matrix_ellpack.M; i++ ){
    for (int j = 0; j < matrix_ellpack.MAXNZ; j++ ){
      JAt[j*matrix_ellpack.M+i] = matrix_ellpack.JA[i*matrix_ellpack.MAXNZ+j];
      AZt[j*matrix_ellpack.M+i] = matrix_ellpack.AZ[i*matrix_ellpack.MAXNZ+j];
    }
  }

  printf("finish loading matrix into 1D tranpose ELLPACK format\n");

  // ======================= Host memory initialisation ======================= //
  
  double* x = (double*) malloc(sizeof(double)*matrix_csr.N);
  double* y_s_c = (double*) malloc(sizeof(double)*matrix_csr.M); //as a reference of result
  double* y_s_e = (double*) malloc(sizeof(double)*matrix_csr.M); // result of serial ellpack 1d
  double* y_c_c = (double*) malloc(sizeof(double)*matrix_csr.M); // result of omp csr
  double* y_c_e1d = (double*) malloc(sizeof(double)*matrix_csr.M); // result of omp ellpack 1d
  double* y_c_e2d = (double*) malloc(sizeof(double)*matrix_csr.M); // result of omp ellpack 2d
  double* y_c_e2dt = (double*) malloc(sizeof(double)*matrix_csr.M); // result of omp ellpack 2d transpose
   
  // random vector element's values
  for (int row = 0; row < matrix_csr.N; ++row) {
    x[row] = 100.0f * ((double) rand()) / RAND_MAX;      
  }
  fprintf(stdout," Matrix-Vector product of %s of size %d x %d\n", matrix_file, matrix_csr.M, matrix_csr.N);

  // ======================= Device memory initialisation ======================= //

  //  Allocate memory space on the device. 
  double *d_csr_AZ, *d_ell_AZ;  // matrix data
  int *d_csr_IRP, *d_csr_JA, *d_ell_JA; // matrix data 
  int *d_ell_JA_2d, *d_ell_JA_2dt;  // 2D ell
  double *d_ell_AZ_2d, *d_ell_AZ_2dt; // 2D ell
  size_t pitch_JA_2d, pitch_AZ_2d, pitch_JA_2dt, pitch_AZ_2dt; // pitch for 2D ell
  double *d_x, *d_y; // vector & result data

  checkCudaErrors(hipMalloc((void**) &d_csr_IRP, (matrix_csr.M+1) * sizeof(int)));
  checkCudaErrors(hipMalloc((void**) &d_csr_JA, matrix_csr.NNZ * sizeof(int)));
  checkCudaErrors(hipMalloc((void**) &d_csr_AZ, matrix_csr.NNZ * sizeof(double)));
  checkCudaErrors(hipMalloc((void**) &d_ell_JA, matrix_csr.M * matrix_ellpack.MAXNZ * sizeof(int)));
  checkCudaErrors(hipMalloc((void**) &d_ell_AZ, matrix_csr.M * matrix_ellpack.MAXNZ * sizeof(double)));
  checkCudaErrors(hipMallocPitch((void**)&d_ell_JA_2d, &pitch_JA_2d, matrix_ellpack.MAXNZ * sizeof(int), matrix_csr.M));
  checkCudaErrors(hipMallocPitch((void**)&d_ell_AZ_2d, &pitch_AZ_2d, matrix_ellpack.MAXNZ * sizeof(double), matrix_csr.M));
  checkCudaErrors(hipMallocPitch((void**)&d_ell_JA_2dt, &pitch_JA_2dt, matrix_csr.M * sizeof(int), matrix_ellpack.MAXNZ));
  checkCudaErrors(hipMallocPitch((void**)&d_ell_AZ_2dt, &pitch_AZ_2dt, matrix_csr.M * sizeof(double), matrix_ellpack.MAXNZ));
  checkCudaErrors(hipMalloc((void**) &d_x, (matrix_csr.N) * sizeof(double)));
  checkCudaErrors(hipMalloc((void**) &d_y, (matrix_csr.M) * sizeof(double)));

  // Copy data from the host (CPU) to the device (GPU).
  checkCudaErrors(hipMemcpy(d_csr_IRP, matrix_csr.IRP, (matrix_csr.M+1) * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_csr_JA, matrix_csr.JA, matrix_csr.NNZ * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_csr_AZ, matrix_csr.AZ, matrix_csr.NNZ * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_ell_JA, matrix_ellpack.JA, matrix_csr.M * matrix_ellpack.MAXNZ * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_ell_AZ, matrix_ellpack.AZ, matrix_csr.M * matrix_ellpack.MAXNZ * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy2D(d_ell_JA_2d, pitch_JA_2d, matrix_ellpack.JA, matrix_ellpack.MAXNZ * sizeof(int), matrix_ellpack.MAXNZ * sizeof(int), matrix_csr.M, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy2D(d_ell_AZ_2d, pitch_AZ_2d, matrix_ellpack.AZ, matrix_ellpack.MAXNZ * sizeof(double), matrix_ellpack.MAXNZ * sizeof(double), matrix_csr.M, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy2D(d_ell_JA_2dt, pitch_JA_2dt, JAt, matrix_csr.M * sizeof(int), matrix_csr.M * sizeof(int), matrix_ellpack.MAXNZ, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy2D(d_ell_AZ_2dt, pitch_AZ_2dt, AZt, matrix_csr.M * sizeof(double), matrix_csr.M * sizeof(double), matrix_ellpack.MAXNZ, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x, x, matrix_csr.N * sizeof(double), hipMemcpyHostToDevice));

  // ======================= Calculations on the CPU ======================= //

  // ----------------------- perform serial code in CSR format ----------------------- //
  timer->reset();
  timer->start();
  for(int tryloop=0; tryloop<ntimes; tryloop++){
    MatrixVectorCSR(matrix_csr.M, matrix_csr.N, matrix_csr.IRP, matrix_csr.JA, matrix_csr.AZ, x, y_s_c);
  }
  timer->stop();

  double time_csr_serial = timer->getTime()/1000/ntimes; // timing
  double mflops_csr_serial = (2.0e-6)*matrix_csr.NNZ/time_csr_serial; // mflops

  fprintf(stdout," [CPU CSR] with 1 thread: time %lf  MFLOPS %lf \n",
	  time_csr_serial,mflops_csr_serial);

  // ----------------------- perform serial code in ELLPACK format ----------------------- //
  timer->reset();
  timer->start();
  for(int tryloop=0; tryloop<ntimes; tryloop++){
    MatrixVectorELLPACK(matrix_ellpack.M, matrix_ellpack.N, matrix_ellpack.NNZ,
     matrix_ellpack.MAXNZ, matrix_ellpack.JA, matrix_ellpack.AZ, x, y_s_e);
  }
  timer->stop();

  double time_ell_serial = timer->getTime()/1000/ntimes;  // timing
  double mflops_ell_serial = (2.0e-6)*matrix_ellpack.NNZ/time_ell_serial; // mflops
  double max_abs_diff_ell_serial, max_rel_diff_ell_serial;
  check_result(matrix_csr.M, y_s_c, y_s_e, &max_abs_diff_ell_serial, &max_rel_diff_ell_serial); // calculate a difference of result

  fprintf(stdout," [CPU ELL] with 1 thread: time %lf  MFLOPS %lf max_abs_diff %lf max_rel_diff %lf\n",
	  time_ell_serial,mflops_ell_serial, max_abs_diff_ell_serial, max_rel_diff_ell_serial);

  // ======================= Calculations on the GPU ======================= //

  // define a 2D block structure
  const dim3 BLOCK_DIM(XBD,YBD);

  // ----------------------- perform parallel code in CSR format ----------------------- //
  // Calculate the dimension of the grid of blocks
  const dim3 GRID_DIM_CSR((matrix_csr.M-1+BLOCK_DIM.y)/BLOCK_DIM.y, 1);

  timer->reset();
  timer->start();
  for(int tryloop=0; tryloop<ntimes; tryloop++){
    gpuMatrixVectorCSR<<<GRID_DIM_CSR, BLOCK_DIM, XBD*YBD*sizeof(double)>>>(BLOCK_DIM.x, BLOCK_DIM.y,
     matrix_csr.M, matrix_csr.N, d_csr_IRP, d_csr_JA, d_csr_AZ, d_x, d_y);
    checkCudaErrors(hipDeviceSynchronize());
  }
  timer->stop();

  // Download the resulting vector d_y from the device and store it in y.
  checkCudaErrors(hipMemcpy(y_c_c, d_y, matrix_csr.M*sizeof(double),hipMemcpyDeviceToHost));
  
  double time_csr_gpu = timer->getTime()/1000/ntimes; // timing
  double mflops_csr_gpu = (2.0e-6)*matrix_csr.NNZ/time_csr_gpu; // mflops
  double max_abs_diff_csr_gpu, max_rel_diff_csr_gpu;
  check_result(matrix_csr.M, y_s_c, y_c_c, &max_abs_diff_csr_gpu, &max_rel_diff_csr_gpu); // calculate a difference of result

  fprintf(stdout," [GPU CSR] Grid dim = %d %d , Block dim = %d %d time %lf  MFLOPS %lf max_abs_diff %lf max_rel_diff %lf\n",
	  GRID_DIM_CSR.x, GRID_DIM_CSR.y, BLOCK_DIM.x, BLOCK_DIM.y, time_csr_gpu,mflops_csr_gpu, max_abs_diff_csr_gpu, max_rel_diff_csr_gpu);

  // ----------------------- perform parallel code in ELLPACK format ----------------------- // 1D //
  // Calculate the dimension of the grid of blocks
  const dim3 GRID_DIM_ELL((matrix_csr.M-1+BLOCK_DIM.y)/BLOCK_DIM.y, 1);

  timer->reset();
  timer->start();
  for(int tryloop=0; tryloop<ntimes; tryloop++){
    gpuMatrixVectorELL<<<GRID_DIM_ELL, BLOCK_DIM, XBD*YBD*sizeof(double)>>>(BLOCK_DIM.x, BLOCK_DIM.y,
     matrix_csr.M, matrix_csr.N, matrix_csr.NNZ, matrix_ellpack.MAXNZ, d_ell_JA, d_ell_AZ, d_x, d_y);
    checkCudaErrors(hipDeviceSynchronize());
  }
  timer->stop();

  // Download the resulting vector d_y from the device and store it in y.
  checkCudaErrors(hipMemcpy(y_c_e1d, d_y, matrix_csr.M*sizeof(double),hipMemcpyDeviceToHost));

  double time_ell_1d_gpu = timer->getTime()/1000/ntimes; // timing
  double mflops_ell_1d_gpu = (2.0e-6)*matrix_csr.NNZ/time_ell_1d_gpu; // mflops
  double max_abs_diff_ell_1d_gpu, max_rel_diff_ell_1d_gpu;
  check_result(matrix_csr.M, y_s_c, y_c_e1d, &max_abs_diff_ell_1d_gpu, &max_rel_diff_ell_1d_gpu); // calculate a difference of result

  fprintf(stdout," [GPU ELL 1D] Grid dim = %d %d , Block dim = %d %d time %lf  MFLOPS %lf max_abs_diff %lf max_rel_diff %lf\n",
	  GRID_DIM_ELL.x, GRID_DIM_ELL.y, BLOCK_DIM.x, BLOCK_DIM.y, time_ell_1d_gpu,mflops_ell_1d_gpu, max_abs_diff_ell_1d_gpu, max_rel_diff_ell_1d_gpu);

  // ----------------------- perform parallel code in ELLPACK format ----------------------- // 2D // * * *

  timer->reset();
  timer->start();
  for(int tryloop=0; tryloop<ntimes; tryloop++){
    gpuMatrixVectorELL_2d<<<GRID_DIM_ELL, BLOCK_DIM, XBD*YBD*sizeof(double)>>>(BLOCK_DIM.x, BLOCK_DIM.y,
     matrix_csr.M, matrix_csr.N, matrix_csr.NNZ, matrix_ellpack.MAXNZ, const_cast<const int*>(d_ell_JA_2d), const_cast<const double*>(d_ell_AZ_2d), d_x, d_y, pitch_JA_2d, pitch_AZ_2d);
    checkCudaErrors(hipDeviceSynchronize());
  }
  timer->stop();

  // Download the resulting vector d_y from the device and store it in y.
  checkCudaErrors(hipMemcpy(y_c_e2d, d_y, matrix_csr.M*sizeof(double),hipMemcpyDeviceToHost));

  double time_ell_2d_gpu = timer->getTime()/1000/ntimes; // timing
  double mflops_ell_2d_gpu = (2.0e-6)*matrix_csr.NNZ/time_ell_2d_gpu; // mflops
  double max_abs_diff_ell_2d_gpu, max_rel_diff_ell_2d_gpu;
  check_result(matrix_csr.M, y_s_c, y_c_e2d, &max_abs_diff_ell_2d_gpu, &max_rel_diff_ell_2d_gpu); // calculate a difference of result

  fprintf(stdout," [GPU ELL 2D] Grid dim = %d %d , Block dim = %d %d time %lf  MFLOPS %lf max_abs_diff %lf max_rel_diff %lf\n",
	  GRID_DIM_ELL.x, GRID_DIM_ELL.y, BLOCK_DIM.x, BLOCK_DIM.y, time_ell_2d_gpu,mflops_ell_2d_gpu, max_abs_diff_ell_2d_gpu, max_rel_diff_ell_2d_gpu);

  // ----------------------- perform parallel code in ELLPACK format ----------------------- // 2D Transpose // * * *

  timer->reset();
  timer->start();
  for(int tryloop=0; tryloop<ntimes; tryloop++){
    gpuMatrixVectorELL_2dt<<<GRID_DIM_ELL, BLOCK_DIM, XBD*YBD*sizeof(double)>>>(BLOCK_DIM.x, BLOCK_DIM.y,
     matrix_csr.M, matrix_csr.N, matrix_csr.NNZ, matrix_ellpack.MAXNZ, const_cast<const int*>(d_ell_JA_2dt), const_cast<const double*>(d_ell_AZ_2dt), d_x, d_y, pitch_JA_2dt, pitch_AZ_2dt);
    checkCudaErrors(hipDeviceSynchronize());
  }
  timer->stop();

  // Download the resulting vector d_y from the device and store it in y.
  checkCudaErrors(hipMemcpy(y_c_e2dt, d_y, matrix_csr.M*sizeof(double),hipMemcpyDeviceToHost));

  double time_ell_2dt_gpu = timer->getTime()/1000/ntimes; // timing
  double mflops_ell_2dt_gpu = (2.0e-6)*matrix_csr.NNZ/time_ell_2dt_gpu; // mflops
  double max_abs_diff_ell_2dt_gpu, max_rel_diff_ell_2dt_gpu;
  check_result(matrix_csr.M, y_s_c, y_c_e2d, &max_abs_diff_ell_2dt_gpu, &max_rel_diff_ell_2dt_gpu); // calculate a difference of result

  fprintf(stdout," [GPU ELL 2DT] Grid dim = %d %d , Block dim = %d %d time %lf  MFLOPS %lf max_abs_diff %lf max_rel_diff %lf\n",
	  GRID_DIM_ELL.x, GRID_DIM_ELL.y, BLOCK_DIM.x, BLOCK_DIM.y, time_ell_2dt_gpu,mflops_ell_2dt_gpu, max_abs_diff_ell_2dt_gpu, max_rel_diff_ell_2dt_gpu);


  // ======================= save result into CSV file ======================= //
  
  save_result_cuda( program_name,      matrix_file,        matrix_csr.M, matrix_csr.N, matrix_csr.NNZ, matrix_ellpack.MAXNZ,
                    BLOCK_DIM.x,       BLOCK_DIM.y,        GRID_DIM_CSR.x,           GRID_DIM_CSR.y,
                    time_csr_serial,   mflops_csr_serial,  0,                        0,
                    time_ell_serial,   mflops_ell_serial,  max_abs_diff_ell_serial,  max_rel_diff_ell_serial,
                    time_csr_gpu,      mflops_csr_gpu,     max_abs_diff_csr_gpu,     max_rel_diff_csr_gpu,
                    time_ell_1d_gpu,   mflops_ell_1d_gpu,  max_abs_diff_ell_1d_gpu,  max_rel_diff_ell_1d_gpu,
                    time_ell_2d_gpu,   mflops_ell_2d_gpu,  max_abs_diff_ell_2d_gpu,  max_rel_diff_ell_2d_gpu,
                    time_ell_2dt_gpu,  mflops_ell_2dt_gpu, max_abs_diff_ell_2dt_gpu, max_rel_diff_ell_2dt_gpu);

  // =======================- Cleaning up ======================= //

  delete timer;

  free(matrix_csr.IRP);
  free(matrix_csr.JA);
  free(matrix_csr.AZ);
  free(matrix_ellpack.JA);
  free(matrix_ellpack.AZ);
  free(JAt);
  free(AZt);
  free(x);
  free(y_s_c);
  free(y_s_e);
  free(y_c_c);
  free(y_c_e1d);
  free(y_c_e2d);
  free(y_c_e2dt);

  checkCudaErrors(hipFree(d_csr_AZ));
  checkCudaErrors(hipFree(d_ell_AZ));
  checkCudaErrors(hipFree(d_ell_AZ_2d));
  checkCudaErrors(hipFree(d_ell_AZ_2dt));
  checkCudaErrors(hipFree(d_csr_IRP));
  checkCudaErrors(hipFree(d_csr_JA));
  checkCudaErrors(hipFree(d_ell_JA));
  checkCudaErrors(hipFree(d_ell_JA_2d));
  checkCudaErrors(hipFree(d_ell_JA_2dt));
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));

  return 0;
}

// ******************** Simple CPU implementation of matrix_vector product multiplication in CSR format ******************** //
void MatrixVectorCSR(int M, int N, const int* IRP, const int* JA,
 const double* AZ, const double* x, double* y) 
{
  int row, col;
  double t;
  for (row = 0; row < M; row++) {
      t = 0;
      for (col = IRP[row]; col < IRP[row+1]; col++) {
          t += AZ[col] * x[JA[col]];
      }
      y[row] = t;
  }
}

// ******************** Simple CPU implementation of matrix_vector product in ELLPACK format ******************** //
void MatrixVectorELLPACK(int M, int N, int NNZ, int MAXNZ, const int* JA,
 const double* AZ, const double* x, double* y) 
{
  int row, col;
  double t;
  int ja_idx;
  for (row = 0; row < M; row++) {
    t = 0;
    for (col = 0; col < MAXNZ; col++) {
      ja_idx = row * MAXNZ + col;
      t += AZ[ja_idx] * x[JA[ja_idx]];
    }
    y[row] = t;
  }
}

// ******************** function to calculate maximum absolute and relative difference of two arrays ******************** //
void check_result(int M, double* y_s_c, double* y, double* max_abs_diff, double* max_rel_diff)
{
  *max_abs_diff = 0;
  *max_rel_diff = 0;

  for(int i=0; i < M; i++){
    double abs_diff = fabs(y_s_c[i] - y[i]);
    *max_abs_diff = fmax(*max_abs_diff, abs_diff);

    double rel_diff = abs_diff / fmax(fabs(y_s_c[i]), fabs(y[i]));
    *max_rel_diff = fmax(*max_rel_diff, rel_diff);
  }
}


// ******************** GPU implementation of matrix_vector product in CSR format ******************** //
__global__ void gpuMatrixVectorCSR(const int XBD, const int YBD, int M, int N, const int* IRP,
 const int* JA, const double* AZ, const double* x, double* y)
{
  int row = blockIdx.x*blockDim.y + threadIdx.y;
  int tid_c = threadIdx.x;
  int tid_r = threadIdx.y;
  int num_threads_per_row = blockDim.x;

  // 1D shared memory is being used because the dimension of the shared memory needs to be specified at runtime.
  extern __shared__ double sdata[]; 

  if (row < M) {
    double t = 0.0;
    for (int col = IRP[row] + tid_c; col < IRP[row+1]; col += blockDim.x) {
      t += AZ[col] * x[JA[col]];
    }
    // Starting address of indexing 1d shared mamory for 2d data
    int sindex = tid_r*XBD+tid_c;
    sdata[sindex] = t;
    __syncthreads();
    
    // Perform row-reduction operation to sum the elements in sdata and store the result in y[row].
    int prev_stride = num_threads_per_row/2;
    for (int stride = num_threads_per_row/2; stride > 0; stride >>= 1) {
      if (tid_c < stride) {
        if(tid_c == stride -1 && prev_stride%2==1){
          sdata[sindex] += sdata[sindex + stride] + sdata[sindex + stride +1];
        }else{
          sdata[sindex] += sdata[sindex + stride];
        }
      }
      __syncthreads();
      prev_stride=stride;
    }

    // Thread 0 writes the final result to global memory
    if (tid_c == 0) {
      y[row] = sdata[sindex];
    }
  }
}

// ******************** GPU implementation of matrix_vector product in ELLPACK format // 1D // ******************** //
__global__ void gpuMatrixVectorELL(const int XBD, const int YBD, int M, int N, int NNZ, int MAXNZ,
 const int* JA, const double* AZ, const double* x, double* y)
{
  int row = blockIdx.x*blockDim.y + threadIdx.y;
  int tid_c = threadIdx.x;
  int tid_r = threadIdx.y;
  int num_threads_per_row = blockDim.x;

  // 1D shared memory is being used because the dimension of the shared memory needs to be specified at runtime.
  extern __shared__ double sdata[];

  if (row < M) {
    double t = 0.0;
    int ja_idx;
    for (int col = tid_c; col < MAXNZ; col += num_threads_per_row) {
      ja_idx = row * MAXNZ + col;
      t += AZ[ja_idx] * x[JA[ja_idx]];
    }
    // Starting address of indexing 1d shared mamory for 2d data
    int sindex = tid_r*XBD+tid_c;
    sdata[sindex] = t;
    __syncthreads();

    // Perform row-reduction operation to sum the elements in sdata and store the result in y[row].
    int prev_stride = num_threads_per_row/2;
    for (int stride = num_threads_per_row/2; stride > 0; stride >>= 1) {
      if (tid_c < stride) {
        if(tid_c == stride -1 && prev_stride%2==1){
          sdata[sindex] += sdata[sindex + stride] + sdata[sindex + stride +1];
        }else{
          sdata[sindex] += sdata[sindex + stride];
        }
      }
      __syncthreads();
      prev_stride=stride;
    }

    // Thread 0 writes the final result to global memory
    if (tid_c == 0) {
      y[row] = sdata[sindex];
    }
  }
}

// ******************** GPU implementation of matrix_vector product in ELLPACK format // 2D // ******************** //
__global__ void gpuMatrixVectorELL_2d(const int XBD, const int YBD, int M, int N, int NNZ, int MAXNZ,
 const int* JA, const double* AZ, const double* x, double* y, size_t pitch_JA, size_t pitch_AZ)
{
  int row = blockIdx.x*blockDim.y + threadIdx.y;
  int tid_c = threadIdx.x;
  int tid_r = threadIdx.y;
  int num_threads_per_row = blockDim.x;

  // 1D shared memory is being used because the dimension of the shared memory needs to be specified at runtime.
  extern __shared__ double sdata[];

  if (row < M) {
    double t1 = 0.0;
    double t2 = 0.0;
    double t3 = 0.0;
    double t4 = 0.0;
    double t5 = 0.0;
    double t6 = 0.0;
    double t7 = 0.0;
    double t8 = 0.0;

    for (int col = tid_c; col < MAXNZ - 7; col += num_threads_per_row * 8) {
      // Compute the address of the (row, col) element in the JA and AZ arrays
      int* row_JA = (int*)((char*)JA + row * pitch_JA);
      double* row_AZ = (double*)((char*)AZ + row * pitch_AZ);

      t1 += row_AZ[col] * x[row_JA[col]];
      t2 += row_AZ[col + 1] * x[row_JA[col + 1]];
      t3 += row_AZ[col + 2] * x[row_JA[col + 2]];
      t4 += row_AZ[col + 3] * x[row_JA[col + 3]];
      t5 += row_AZ[col + 4] * x[row_JA[col + 4]];
      t6 += row_AZ[col + 5] * x[row_JA[col + 5]];
      t7 += row_AZ[col + 6] * x[row_JA[col + 6]];
      t8 += row_AZ[col + 7] * x[row_JA[col + 7]];
    }

    double t = t1 + t2 + t3 + t4 + t5 + t6 + t7 + t8;

    // Store result in shared memory
    int sindex = tid_r * XBD + tid_c;
    sdata[sindex] = t;
    __syncthreads();

    // Perform row-reduction operation to sum the elements in sdata and store the result in y[row].
    int prev_stride = num_threads_per_row/2;
    for (int stride = num_threads_per_row/2; stride > 0; stride >>= 1) {
      if (tid_c < stride) {
        if(tid_c == stride -1 && prev_stride%2==1){
          sdata[sindex] += sdata[sindex + stride] + sdata[sindex + stride +1];
        }else{
          sdata[sindex] += sdata[sindex + stride];
        }
      }
      __syncthreads();
      prev_stride=stride;
    }

    // Thread 0 writes the final result to global memory
    if (tid_c == 0) {
      y[row] = sdata[sindex];
    }
  }
}

// ******************** function to save result into CSV file ******************** //
void save_result_cuda(char *program_name,      char* matrix_file,          int M, int N,                     int NNZ, int MAZNZ,
                      int cudaXBD,             int cudaYBD,                int cudaXGD,                      int cudaYGD,
                      double time_csr_serial,  double mflops_csr_serial,   double max_abs_diff_csr_serial,   double max_rel_diff_csr_serial,
                      double time_ell_serial,  double mflops_ell_serial,   double max_abs_diff_ell_serial,   double max_rel_diff_ell_serial,
                      double time_csr_gpu,     double mflops_csr_gpu,      double max_abs_diff_csr_gpu,      double max_rel_diff_csr_gpu,
                      double time_ell_1d_gpu,  double mflops_ell_1d_gpu,   double max_abs_diff_ell_1d_gpu,   double max_rel_diff_ell_1d_gpu,
                      double time_ell_2d_gpu,  double mflops_ell_2d_gpu,   double max_abs_diff_ell_2d_gpu,   double max_rel_diff_ell_2d_gpu, 
                      double time_ell_2dt_gpu, double mflops_ell_2dt_gpu,  double max_abs_diff_ell_2dt_gpu,  double max_rel_diff_ell_2dt_gpu)
{
  // open file for appending or create new file with header
  FILE *fp;
  char filename[100];
  strcpy(filename, default_filename);
  fp = fopen(filename, "a+");
  if (fp == NULL) {
    printf("Error opening file.\n");
    exit(1);
  }
  // check if file is empty
  fseek(fp, 0, SEEK_END);
  long file_size = ftell(fp);
  if (file_size == 0) {
    // add header row
    fprintf(fp, "program_name,matrix_file,M,N,NNZ,MAXNZ,");
    fprintf(fp, "cudaXBD,cudaYBD,cudaXGD,cudaYGD,");
    fprintf(fp, "time_csr_serial,mflops_csr_serial,max_abs_diff_csr_serial,max_rel_diff_csr_serial,");
    fprintf(fp, "time_ell_serial,mflops_ell_serial,max_abs_diff_ell_serial,max_rel_diff_ell_serial,");
    fprintf(fp, "time_csr_gpu,mflops_csr_gpu,max_abs_diff_csr_gpu,max_rel_diff_csr_gpu,");
    fprintf(fp, "time_ell_1d_gpu,mflops_ell_1d_gpu,max_abs_diff_ell_1d_gpu,max_rel_diff_ell_1d_gpu,");
    fprintf(fp, "time_ell_2d_gpu,mflops_ell_2d_gpu,max_abs_diff_ell_2d_gpu,max_rel_diff_ell_2d_gpu,");
    fprintf(fp, "time_ell_2dt_gpu,mflops_ell_2dt_gpu,max_abs_diff_ell_2dt_gpu,max_rel_diff_ell_2dt_gpu\n");
  }

  // write new row to file
  fprintf(fp, "%s,%s,%d,%d,%d,%d,%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",
          program_name,      matrix_file,        M, N,                      NNZ, MAZNZ,
          cudaXBD,           cudaYBD,            cudaXGD,                   cudaYGD,
          time_csr_serial,   mflops_csr_serial,  max_abs_diff_csr_serial,   max_rel_diff_csr_serial,
          time_ell_serial,   mflops_ell_serial,  max_abs_diff_ell_serial,   max_rel_diff_ell_serial,
          time_csr_gpu,      mflops_csr_gpu,     max_abs_diff_csr_gpu,      max_rel_diff_csr_gpu,
          time_ell_1d_gpu,   mflops_ell_1d_gpu,  max_abs_diff_ell_1d_gpu,   max_rel_diff_ell_1d_gpu,
          time_ell_2d_gpu,   mflops_ell_2d_gpu,  max_abs_diff_ell_2d_gpu,   max_rel_diff_ell_2d_gpu,
          time_ell_2dt_gpu,  mflops_ell_2dt_gpu, max_abs_diff_ell_2dt_gpu,  max_rel_diff_ell_2dt_gpu);

  // close file
  fclose(fp);
}