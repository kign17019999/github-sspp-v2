#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers

//Simple dimension: define a 1D block structure
#define BD 256

const dim3 BLOCK_DIM(BD);


#include <stdlib.h>
#include <stdio.h>
#include "wtime.h"
#include "read_csr.h"
#include "read_ellpack.h"
inline double dmin ( double a, double b ) { return a < b ? a : b; }

void MatrixVectorCSR(int M, int N, const int* IRP, const int* JA,
 const double* AZ, const double* x, double* y);
void printCSR(int M, int N, int NNZ, const int* IRP, const int* JA,
 const double* AZ);
void MatrixVectorELLPACK(int M, int N, int NNZ, int MAXNZ, const int* JA,
 const double* AZ, const double* x, double* y);
void printELLPACK(int M, int N, int NNZ, int MAXNZ, const int* JA,
 const double* AZ);
int check_result(int M, double* y0, double* y);

__global__ void gpuMatrixVectorCSR(int M, int N, const int* IRP, const int* JA,
 const double* AZ, const double* x, double* y);
__global__ void gpuMatrixVectorELL(int M, int N, int NNZ, int MAXNZ, const int* JA,
 const double* AZ, const double* x, double* y);


int main(int argc, char** argv) 
{
  // Create the CUDA SDK timer.
  StopWatchInterface* timer = 0;
  sdkCreateTimer(&timer);
  timer->reset();

  printf("run from file %s\n", argv[0]);
  char* matrix_file = "matrices/cage4.mtx"; // set default file name
  if (argc == 2) {
    matrix_file = argv[1];
  } else {
    printf("Usage: main <matrix_file> \n");
    return -1;
  }

  /* reading file into CSR */
  struct csr_matrix matrix_csr;
  int ret_code;
  ret_code = read_csr_matrix(matrix_file, &matrix_csr);
  if (ret_code != 0) {
    printf("Failed to read matrix file\n");
    return ret_code;
  }
  //printCSR(matrix_csr.M, matrix_csr.N, matrix_csr.NNZ, matrix_csr.IRP, matrix_csr.JA, matrix_csr.AZ);
  /* END reading file into CSR */

  /* reading file into ELLPACK */
  struct ellpack_matrix matrix_ellpack;
  ret_code = read_ellpack_matrix(matrix_file, &matrix_ellpack);
  if (ret_code != 0) {
    printf("Failed to read matrix file\n");
    return ret_code;
  }
  //printELLPACK(matrix_ellpack.M, matrix_ellpack.N, matrix_ellpack.NNZ, matrix_ellpack.MAXNZ, matrix_ellpack.JA, matrix_ellpack.AZ);
  /* END reading file into ELLPACK */

  double* x = (double*) malloc(sizeof(double)*matrix_csr.N);
  double* y = (double*) malloc(sizeof(double)*matrix_csr.M);
  double* y0 = (double*) malloc(sizeof(double)*matrix_csr.M);
  int row;
  for ( row = 0; row < matrix_csr.M; ++row) {
    x[row] = 100.0f * ((double) rand()) / RAND_MAX;      
  }
  double t1, t2;
  fprintf(stdout,"Matrix-Vector product of %s of size %d x %d\n", matrix_file, matrix_csr.M, matrix_csr.N);
  
  /* CSR Serial*/
  double tmlt_csr_serial = 1e100;
  timer->start();
  t1 = wtime();
  MatrixVectorCSR(matrix_csr.M, matrix_csr.N, matrix_csr.IRP, matrix_csr.JA,
   matrix_csr.AZ, x, y0);
  t2 = wtime();
  timer->stop();
  tmlt_csr_serial = dmin(tmlt_csr_serial,(t2-t1));
  double mflops_csr_serial = (2.0e-6)*matrix_csr.NNZ/tmlt_csr_serial;
  fprintf(stdout,"[CSR] with 1 thread: time %lf  MFLOPS %lf \n",
	  tmlt_csr_serial,mflops_csr_serial);

  double mflops_csr_serial2 = (2.0e-6)*matrix_csr.NNZ/(timer->getTime()/1000);
  fprintf(stdout,"[CSR 2] with X thread: time %lf  MFLOPS %lf\n",
	  timer->getTime(),mflops_csr_serial2);
  
  /* END CSR Serial */

  /* ELLPACK Serial */
  double tmlt_ell_serial = 1e100;
  timer->start();
  t1 = wtime();
  MatrixVectorELLPACK(matrix_ellpack.M, matrix_ellpack.N, matrix_ellpack.NNZ,
   matrix_ellpack.MAXNZ, matrix_ellpack.JA, matrix_ellpack.AZ, x, y);
  t2 = wtime();
  timer->stop();
  tmlt_ell_serial = dmin(tmlt_ell_serial,(t2-t1));
  double mflops_ell_serial = (2.0e-6)*matrix_ellpack.NNZ/tmlt_ell_serial;
  double max_diff_ell_serial = check_result(matrix_csr.M, y0, y);
  fprintf(stdout,"[ELL] with 1 thread: time %lf  MFLOPS %lf max_diff %lf\n",
	  tmlt_ell_serial,mflops_ell_serial, max_diff_ell_serial);

  double mflops_ell_serial2 = (2.0e-6)*matrix_csr.NNZ/(timer->getTime()/1000);
  fprintf(stdout,"[ELL 2] with X thread: time %lf  MFLOPS %lf max_diff %lf\n",
	  timer->getTime(),mflops_ell_serial2, max_diff_csr_cuda); 
  /* END ELLPACK Serial */

  /* ================================== */

  int *d_M, *d_N, *d_NNZ;
  int *d_ell_MAXNZ;
  checkCudaErrors(hipMalloc(&d_M, sizeof(int)));
  checkCudaErrors(hipMalloc(&d_N, sizeof(int)));
  checkCudaErrors(hipMalloc(&d_NNZ, sizeof(int)));
  checkCudaErrors(hipMalloc(&d_ell_MAXNZ, sizeof(int)));

  checkCudaErrors(hipMemcpy(d_M, &matrix_csr.M, sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_N, &matrix_csr.N, sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_NNZ, &matrix_csr.NNZ, sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_ell_MAXNZ, &matrix_ellpack.MAXNZ, sizeof(int), hipMemcpyHostToDevice));

  double *d_csr_AZ, *d_ell_AZ;
  int *d_csr_IRP, *d_csr_JA, *d_ell_JA;
  checkCudaErrors(hipMalloc((void**) &d_csr_IRP, (matrix_csr.M+1) * sizeof(int)));
  checkCudaErrors(hipMalloc((void**) &d_csr_JA, matrix_csr.NNZ * sizeof(int)));
  checkCudaErrors(hipMalloc((void**) &d_csr_AZ, matrix_csr.NNZ * sizeof(double)));
  checkCudaErrors(hipMalloc((void**) &d_ell_JA, matrix_csr.M * matrix_ellpack.MAXNZ * sizeof(int)));
  checkCudaErrors(hipMalloc((void**) &d_ell_AZ, matrix_csr.M * matrix_ellpack.MAXNZ * sizeof(double)));

  checkCudaErrors(hipMemcpy(d_csr_IRP, matrix_csr.IRP, (matrix_csr.M+1) * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_csr_JA, matrix_csr.JA, matrix_csr.NNZ * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_csr_AZ, matrix_csr.AZ, matrix_csr.NNZ * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_ell_JA, matrix_ellpack.JA, matrix_csr.M * matrix_ellpack.MAXNZ * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_ell_AZ, matrix_ellpack.AZ, matrix_csr.M * matrix_ellpack.MAXNZ * sizeof(double), hipMemcpyHostToDevice));

  double *d_x, *d_y;
  checkCudaErrors(hipMalloc((void**) &d_x, (matrix_csr.N) * sizeof(double)));
  checkCudaErrors(hipMalloc((void**) &d_y, (matrix_csr.M) * sizeof(double)));
  checkCudaErrors(hipMemcpy(d_x, x, matrix_csr.N * sizeof(double), hipMemcpyHostToDevice));

  const dim3 GRID_DIM((matrix_csr.M - 1 + BLOCK_DIM.x)/ BLOCK_DIM.x  ,1);
  printf("grid dim = %d , block dim = %d \n",GRID_DIM.x,BLOCK_DIM.x);

  timer->start();
  gpuMatrixVectorCSR<<<GRID_DIM, BLOCK_DIM >>>(matrix_csr.M, matrix_csr.N, d_csr_IRP, d_csr_JA, d_csr_AZ, d_x, d_y);
  checkCudaErrors(hipDeviceSynchronize());
  timer->stop();
  checkCudaErrors(hipMemcpy(y, d_y, matrix_csr.N*sizeof(double),hipMemcpyDeviceToHost));
  double mflops_csr_cuda = (2.0e-6)*matrix_csr.NNZ/(timer->getTime()/1000);
  double max_diff_csr_cuda = check_result(matrix_csr.M, y0, y);
  fprintf(stdout,"[CSR cuda] with X thread: time %lf  MFLOPS %lf max_diff %lf\n",
	  timer->getTime(),mflops_csr_cuda, max_diff_csr_cuda);

  timer->reset();
  timer->start();
  gpuMatrixVectorELL<<<GRID_DIM, BLOCK_DIM >>>(matrix_csr.M, matrix_csr.N, matrix_csr.NNZ, matrix_ellpack.MAXNZ, d_ell_JA, d_ell_AZ, d_x, d_y);
  checkCudaErrors(hipDeviceSynchronize());
  timer->stop();
  checkCudaErrors(hipMemcpy(y, d_y, matrix_csr.N*sizeof(double),hipMemcpyDeviceToHost));
  double mflops_ell_cuda = (2.0e-6)*matrix_csr.NNZ/(timer->getTime()/1000);
  double max_diff_ell_cuda = check_result(matrix_csr.M, y0, y);
  fprintf(stdout,"[ELL cuda] with X thread: time %lf  MFLOPS %lf max_diff %lf\n",
	  timer->getTime(),mflops_ell_cuda, max_diff_ell_cuda);

  /* ================================== */

  free(matrix_csr.IRP);
  free(matrix_csr.JA);
  free(matrix_csr.AZ);
  free(matrix_ellpack.JA);
  free(matrix_ellpack.AZ);
  free(x);
  free(y);
  free(y0);
  return 0;
}

void MatrixVectorCSR(int M, int N, const int* IRP, const int* JA,
 const double* AZ, const double* x, double* y) 
{
  int row, col;
  double t;
  for (row = 0; row < M; row++) {
      t = 0;
      for (col = IRP[row]; col < IRP[row+1]; col++) {
          t += AZ[col] * x[JA[col]];
      }
      y[row] = t;
  }
}

void printCSR(int M, int N, int NNZ, const int* IRP, const int* JA,
 const double* AZ){
  printf("CSR representation:\n");
  printf("M: %d\nN: %d\n", M, N);
  printf("NNZ: %d\n", NNZ);
  printf("IRP: ");
  int i;
  for (i = 0; i < M + 1; i++) {
    printf("%d ", IRP[i]);
    if(i!=M-1 && i==5 && M>11){
      printf("... ");
      i=M-5;
    }
  }
  printf("\nJA: ");
  for (i = 0; i < NNZ; i++) {
    printf("%d ", JA[i]);
    if(i!=NNZ-1 && i==5 && NNZ>11){
      printf("... ");
      i=NNZ-5;
    }
  }
  printf("\nAZ: ");
  for (i = 0; i < NNZ; i++) {
    printf("%.3lf ", AZ[i]);
    if(i!=NNZ-1 && i==5 && NNZ>11){
      printf("... ");
      i=NNZ-5;
    }
  }
  printf("\n");
}

void MatrixVectorELLPACK(int M, int N, int NNZ, int MAXNZ, const int* JA,
 const double* AZ, const double* x, double* y) 
{
  int row, col;
  double t;
  for (row = 0; row < M; row++) {
    t = 0;
    for (col = 0; col < MAXNZ; col++) {
      int ja_idx = row * MAXNZ + col;
      if (col >= NNZ || JA[ja_idx] < 0) {
        break;
      }
      t += AZ[ja_idx] * x[JA[ja_idx]];
    }
    y[row] = t;
  }
}

void printELLPACK(int M, int N, int NNZ, int MAXNZ, const int* JA,
 const double* AZ){
  printf("ELLPACK representation:\n");
  printf("M: %d\nN: %d\n", M, N);
  printf("NNZ: %d\n", NNZ);
  printf("MAXNZ: %d\n", MAXNZ);
  printf("JA: \n");
  int i, j;
  for (i = 0; i < M; i++) {
    for (j = 0; j < MAXNZ; j++) {
      printf("%d ", JA[i * MAXNZ + j]);
      if(j!=MAXNZ-1 && j==5 && MAXNZ>11){
        printf("... ");
        j=MAXNZ-5;
      }
    }
    printf("\n");
    if(i!=M-1 && i==5 && M>11){
      printf(" ... \n");
      i=M-5;
    }
  }
  printf("AZ: \n");
  for (i = 0; i < M; i++) {
    for (j = 0; j < MAXNZ; j++) {
      printf("%.3lf ", AZ[i * MAXNZ + j]);
      if(j!=MAXNZ-1 && j==5 && MAXNZ>11){
        printf("... ");
        j=MAXNZ-5;
      }
    }
    printf("\n");
    if(i!=M-1 && i==5 && M>11){
      printf(" ... \n");
      i=M-5;
    }
  }
}

int check_result(int M, double* y0, double* y)
{
  double max_diff = 0;
  double cal_diff = 0;
  for(int i=0; i < M; i++){
    cal_diff = abs(y0[i] - y[i]);
    if(max_diff < cal_diff) max_diff = cal_diff;
  }
  return max_diff;
}

__global__ void gpuMatrixVectorCSR(int M, int N, const int* IRP, const int* JA,
 const double* AZ, const double* x, double* y)
{
  int tr = threadIdx.x;
  int row = blockIdx.x*blockDim.x + tr;
  if (row < M) {
    double t = 0;
    for (int col = IRP[row]; col < IRP[row+1]; col++) {
      t += AZ[col] * x[JA[col]];
    }
    y[row] = t;
  }
}

__global__ void gpuMatrixVectorELL(int M, int N, int NNZ, int MAXNZ, const int* JA,
 const double* AZ, const double* x, double* y)
{
  int tr = threadIdx.x;
  int row = blockIdx.x*blockDim.x + tr;
  if (row < M) {
    double t = 0;
    for (int col = 0; col < MAXNZ; col++) {
      int ja_idx = row * MAXNZ + col;
      if (col >= NNZ || JA[ja_idx] < 0) {
        break;
      }
      t += AZ[ja_idx] * x[JA[ja_idx]];
    }
    y[row] = t;
  }
}